// nvcc -Xcompiler -Wall -DDOLOG ppm-cuda.cu

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cstdio>
#include "cuda_helpers.h"

int main(int argc, char *argv[])
{
    int width, height;

    // these are the vectors storing the pixeldata in its three colors
    std::vector<float> r,g,b;
    float *d_r, *d_g, *d_b, *h_r, *h_g, *h_b;

    // read file
    read("input.ppm", width, height, r,g,b);
    LOG("processing " << width << "x" << height);

    // allocate memory for destination and "copy"
    gpuErrchk(hipMalloc(&d_r, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&d_g, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&h_r, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&h_g, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&h_b, width*height*sizeof(float)));

    //set i <= 5 or more for blur() testing
    for (int i = 0; i <= 1; i++) {

      // copy to GPU
      gpuErrchk(hipMemcpy(d_r, &r[0], width*height*sizeof(float), hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(d_g, &g[0], width*height*sizeof(float), hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(d_b, &b[0], width*height*sizeof(float), hipMemcpyHostToDevice));

      // call kernel on GPU
      dim3 tpb(16, 16);
      dim3 bpg((width+tpb.x-1)/tpb.x, (height+tpb.y-1)/tpb.y);

      //uncomment for blur()
      //blur<<<bpg,tpb>>>(width, height, d_r, d_g, d_b, h_r, h_g, h_b, 0.25);

      //uncomment for edgedetect()
      //edgedetect<<<bpg,tpb>>>(width, height, d_r, d_g, d_b, h_r, h_g, h_b, 0.2);

      //uncomment for sinewave()
      //sinewave<<<bpg,tpb>>>(width, height, h_r, h_g, h_b);

      //uncomment for flippy()
      flippy<<<bpg,tpb>>>(width, height, d_r, d_g, d_b, h_r, h_g, h_b);

      // prep arrays for receipt from kernel
      r.resize(width*height);
      g.resize(width*height);
      b.resize(width*height);

      // copy data back from GPU
      gpuErrchk(hipMemcpy(&r[0], h_r, width*height*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&g[0], h_g, width*height*sizeof(float), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&b[0], h_b, width*height*sizeof(float), hipMemcpyDeviceToHost));

    }

    // be a good roommate and clean up
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(h_r);
    hipFree(h_g);
    hipFree(h_b);

    // save image
    write("output.ppm", width, height, r,g,b);

    return 0;
}
