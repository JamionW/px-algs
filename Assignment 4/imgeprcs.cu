#include "hip/hip_runtime.h"
//  Dr. Tanis did most of the hard work...
//  ...thanks Dr. Tanis!
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cstdio>
#include "cuda_helpers.h"

// read file courtesy of Dr. Tanis
char* data;
int read(std::string filename,
         int& width,
         int& height,
         std::vector<float>& r,
         std::vector<float>& g,
         std::vector<float>& b)
{
    std::ifstream in(filename.c_str(), std::ios::binary);

    int maxcol;

    if (! in.is_open())
    {
        std::cerr << "could not open " << filename << " for reading" << std::endl;
        return 0;
    }

    {
        std::string magicNum;
        in >> magicNum;
        LOG("got magicNum:" << magicNum);
        //JW what is this?
        // this is broken if magicNum != 'P6'
    }

    {
        long loc = in.tellg();
        std::string comment;
        in >> comment;

        if (comment[0] != '#')
        {
            in.seekg(loc);
        }
        else
        {
            LOG("got comment:" << comment);
        }
    }

    in >> width >> height >> maxcol;
    in.get();                   // eat newline
    LOG("dimensions: " << width << "x" << height << "("<<maxcol<<")");


    data = new char[width*height*3];
    in.read(data, width*height*3);
    in.close();

    r.resize(width*height);
    g.resize(width*height);
    b.resize(width*height);

    for (int i=0; i<width*height; ++i)
    {
        int base = i*3;
        r[i] =  ((unsigned char)data[base+0])/255.0f;
        g[i] =  ((unsigned char)data[base+1])/255.0f;
        b[i] =  ((unsigned char)data[base+2])/255.0f;
    }
    free(data);

    return 1;
}

// write couresy of Dr. Tanis
int write(std::string outfile,
          int width, int height,
          const std::vector<float>& r,
          const std::vector<float>& g,
          const std::vector<float>& b)
{
    std::ofstream ofs(outfile.c_str(), std::ios::out | std::ios::binary);

    if (! ofs.is_open())
    {
        std::cerr << "could not open " << outfile << " for writing" << std::endl;
    }

    ofs << "P6\n#*\n" << width << " " << height << "\n255\n";

    for (int i=0; i < width*height; ++i)
    {
        ofs <<
            (unsigned char)(r[i]*255) <<
            (unsigned char)(g[i]*255) <<
            (unsigned char)(b[i]*255);
    }
    ofs.close();

    return 1;
}

//blur courtesy of Dr. Tanis
__global__
void blur(int width, int height,
          float* r, float* g, float* b,
          float* r_o, float* g_o, float* b_o,
          float weight)
{
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;

    if (x_idx < width && y_idx < height)
    {

        // active CUDA thread

        int s_idx = x_idx + y_idx*width;

        int nnbr = 0;
        float sum_r=0, sum_g=0, sum_b=0;

        if (x_idx > 0)
        {
            // left
            nnbr++;
            int left = s_idx-1;

            sum_r += r[left];
            sum_g += g[left];
            sum_b += b[left];
        }

        if (x_idx < width-1)
        {
            //right
            nnbr++;
            int right = s_idx+1;
            sum_r += r[right];
            sum_g += g[right];
            sum_b += b[right];
        }

        if (y_idx > 0)
        {
            //up
            nnbr++;
            int up = s_idx - width;
            sum_r += r[up];
            sum_g += g[up];
            sum_b += b[up];
        }

        if (y_idx < height-1)
        {
            //up
            nnbr++;
            int down = s_idx + width;
            sum_r += r[down];
            sum_g += g[down];
            sum_b += b[down];
        }

        if (nnbr > 0)
        {
            r_o[s_idx] = r[s_idx]*weight + sum_r * (1-weight) / nnbr;
            g_o[s_idx] = g[s_idx]*weight + sum_g * (1-weight) / nnbr;
            b_o[s_idx] = b[s_idx]*weight + sum_b * (1-weight) / nnbr;
        }
        else
        {
            r_o[s_idx] = r[s_idx];
            g_o[s_idx] = g[s_idx];
            b_o[s_idx] = b[s_idx];
        }
    }
}

__global__
void edgedetect(int width, int height,
          float* r, float* g, float* b,
          float* r_o, float* g_o, float* b_o,
          float threshold)
{
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;

    if (x_idx < width && y_idx < height)
    {
        int s_idx = x_idx + y_idx*width;
        int nnbr = 0, d_x = 0, d_y = 0;
        float diff = 0, sum_left = 0, sum_right = 0, sum_up = 0, sum_down = 0;

        if (x_idx > 0)
        {
            // left
            nnbr++;
            int left = s_idx-1;
            sum_left += r[left] + g[left] + b[left];
        }

        if (x_idx < width-1)
        {
            //right
            nnbr++;
            int right = s_idx+1;
            sum_right += r[right] + g[right] + b[right];
        }

        if (y_idx > 0)
        {
            //up
            nnbr++;
            int up = s_idx - width;
            sum_up += r[up] + g[up] + b[up];
        }

        if (y_idx < height-1)
        {
            //down
            nnbr++;
            int down = s_idx + width;
            sum_down += r[down] + g[down] + b[down];
        }


        if (nnbr > 0)
        {
            d_x = (-0.5 * sum_left) + (-0.5 * sum_right);
            d_y = (-0.5 * sum_down) + (-0.5 * sum_up);
            diff = sqrtf((powf(d_x, 2)) + (powf(d_y, 2)));

            if (diff >= threshold) {
              r_o[s_idx] = 1;
              g_o[s_idx] = 1;
              b_o[s_idx] = 1;
            }
            else
            {
              r_o[s_idx] = 0;
              g_o[s_idx] = 0;
              b_o[s_idx] = 0;
            }
        }
        else
        {
            r_o[s_idx] = r[s_idx];
            g_o[s_idx] = g[s_idx];
            b_o[s_idx] = b[s_idx];
        }
    }
}

__global__
void sinewave(int width, int height,
          float* r_o, float* g_o, float* b_o)
{
  int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
  float radian = (x_idx) * (3.14/180);
  float sineVal = sin(radian);

  if (x_idx < width && y_idx < height)
  {
      int s_idx = x_idx + y_idx*width;
      float sineAlter = ((sineVal * x_idx) + (height / 4));
      int numSin = (int)sineAlter;

        if (y_idx >= sineAlter) {
          r_o[s_idx] = 0;
          g_o[s_idx] = sineAlter - numSin;
          b_o[s_idx] = sineAlter - numSin / 2;
        }
        else
        {
          r_o[s_idx] = 0.75;
          g_o[s_idx] = 0.75;
          b_o[s_idx] = 0.75;
        }
  }
}


__global__
void flippy(int width, int height,
          float* r, float* g, float* b,
          float* r_o, float* g_o, float* b_o)
{
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int s_idx = x_idx + y_idx * width;
    int xdi_s = -x_idx + (-1 * (y_idx)) * width;
    //int half = (width * height) / 2;

    if (x_idx < width && y_idx < height)
    //if (s_idx >= half)
    {
        r_o[xdi_s] = r[s_idx];
        g_o[xdi_s] = g[s_idx];
        b_o[xdi_s] = b[s_idx];
    }
}
